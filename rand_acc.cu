/* 
    Copyright (C) 2016  N. Perna, N. Nedialkov, T. Gwosdz
  
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "a3.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//#ifndef MY_DUMMY
//#define MY_DUMMY
extern "C"{
unsigned int  randPlz();
}
//#endif


extern "C"{
#pragma acc routine seq
unsigned int randPlz () 
{
  unsigned int* rand_buffer = NULL;
      hipMalloc((void **) &rand_buffer, 1*sizeof(unsigned int));
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
      hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
      hiprandGenerate(gen, rand_buffer, 1);
      printf("sdsdsd = %u\n", rand_buffer);
      long int r = *rand_buffer;
  return r;//hiprand();
}
}
